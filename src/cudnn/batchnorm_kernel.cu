/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

void BatchNorm::map(void)
{
  assert(inputs[0].numDim == 4);
  // create descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  int inputN = inputs[0].dim[0];
  int inputC = inputs[0].dim[1];
  int inputH = inputs[0].dim[2];
  int inputW = inputs[0].dim[3];
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, inputN, inputC, inputH, inputW));
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, inputN, inputC, inputH, inputW));
  checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, 1, inputC, 1, 1));
#ifdef DO_TRAINING
  checkCUDA(hipMalloc(&runningMean, sizeof(DATATYPE) * inputC));
  checkCUDA(hipMalloc(&runningVar, sizeof(DATATYPE) * inputC));
  checkCUDA(hipMalloc(&saveMean, sizeof(DATATYPE) * inputC));
  checkCUDA(hipMalloc(&saveVar, sizeof(DATATYPE) * inputC));
  checkCUDA(hipMalloc(&biasPtr, sizeof(DATATYPE) * inputC));
  checkCUDA(hipMalloc(&scalePtr, sizeof(DATATYPE) * inputC));
   initialize scale to ones and bias to zeros
  assign_kernel<<<GET_BLOCKS(inputC), CUDA_NUM_THREADS>>>(
    scalePtr, inputC, 1.0f);
  assign_kernel<<<GET_BLOCKS(inputC), CUDA_NUM_THREADS>>>(
    biasPtr, inputC, 0.0f);
  assign_kernel<<<GET_BLOCKS(inputC), CUDA_NUM_THREADS>>>(
    runningMean, inputC, 0.0f);
  assign_kernel<<<GET_BLOCKS(inputC), CUDA_NUM_THREADS>>>(
    runningVar, inputC, 0.0f);
#endif
  size_t outputSize = sizeof(DATATYPE) * outputs[0].volume();
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputSize));
}

void BatchNorm::unmap(void)
{
  checkCUDNN(hipdnnDestroyTensorDescriptor(inputTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(biasTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensor));
#ifdef DO_TRAINING
  checkCUDA(hipFree(runningMean));
  checkCUDA(hipFree(runningVar));
  checkCUDA(hipFree(saveMean));
  checkCUDA(hipFree(saveVar));
  checkCUDA(hipFree(biasPtr));
  checkCUDA(hipFree(scalePtr));
  checkCUDA(hipFree(outputs[0].data_ptr));
#endif
}

void BatchNorm::forward(bool block)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipdnnBatchNormMode_t mode = HIPDNN_BATCHNORM_SPATIAL;
  //int inputC = inputs[0].dim[1];
#ifdef DO_TRAINING 
  if (model->isTraining) {
    assign_kernel<<<GET_BLOCKS(inputC), CUDA_NUM_THREADS>>>(
      runningMean, inputC, 0.0f);
    assign_kernel<<<GET_BLOCKS(inputC), CUDA_NUM_THREADS>>>(
      runningVar, inputC, 0.0f);
    checkCUDNN(hipdnnBatchNormalizationForwardTraining(
      model->dnn, mode, &alpha, &beta, inputTensor, inputs[0].data_ptr,
      outputTensor, outputs[0].data_ptr, biasTensor, scalePtr, biasPtr,
      1.0, runningMean, runningVar, HIPDNN_BN_MIN_EPSILON, saveMean, saveVar));
  } else {
#endif
    checkCUDNN(hipdnnBatchNormalizationForwardInference(
      model->dnn, mode, &alpha, &beta, inputTensor, inputs[0].data_ptr,
      outputTensor, outputs[0].data_ptr, biasTensor, inputs[1].data_ptr, inputs[2].data_ptr,
      inputs[3].data_ptr, inputs[4].data_ptr, HIPDNN_BN_MIN_EPSILON)); 
#ifdef DO_TRAINING 
  }
#endif
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_batchnorm_cost(BatchNorm* bn)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;
  int inputC = bn->inputs[0].dim[1];
  int inputH = bn->inputs[0].dim[2];
  int inputW = bn->inputs[0].dim[3];
  hipdnnBatchNormMode_t mode = HIPDNN_BATCHNORM_SPATIAL;
  checkCUDNN(hipdnnSetTensor4dDescriptor(inputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, BATCH_SIZE, inputC, inputH, inputW));
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, BATCH_SIZE, inputC, inputH, inputW));
  checkCUDNN(hipdnnSetTensor4dDescriptor(biasTensor, HIPDNN_TENSOR_NCHW,
      HIPDNN_DATA_FLOAT, 1, inputC, 1, 1));
#ifdef DO_TRAINING
  assign_kernel<<<GET_BLOCKS(inputC), CUDA_NUM_THREADS>>>(
    scalePtr, inputC, 0.5f);
  assign_kernel<<<GET_BLOCKS(inputC), CUDA_NUM_THREADS>>>(
    biasPtr, inputC, 0.5f);
#endif
  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int i = 0; i < REPEAT_TIMES; i++) {
#ifdef DO_TRAINING
    if (isTraining) {
      assign_kernel<<<GET_BLOCKS(inputC), CUDA_NUM_THREADS>>>(
        runningMean, inputC, 0.0f);
      assign_kernel<<<GET_BLOCKS(inputC), CUDA_NUM_THREADS>>>(
        runningVar, inputC, 0.0f);
      checkCUDNN(hipdnnBatchNormalizationForwardTraining(
        dnn, mode, &alpha, &beta, inputTensor, inputPtr,
        outputTensor, outputPtr, biasTensor, scalePtr, biasPtr,
        1.0, runningMean, runningVar, HIPDNN_BN_MIN_EPSILON,
        saveMean, saveVar));
    } else {
#endif
      checkCUDNN(hipdnnBatchNormalizationForwardInference(
        dnn, mode, &alpha, &beta, inputTensor, inputPtr,
        outputTensor, outputPtr, biasTensor, scalePtr, biasPtr,
        runningMean, runningVar, HIPDNN_BN_MIN_EPSILON));
#ifdef DO_TRAINING
    }
#endif
  }
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  float milliseconds;
  hipEventElapsedTime(&milliseconds, startEvent, endEvent);
  bn->runtime = milliseconds / REPEAT_TIMES;
  printf("measure[BatchNorm]: i(%d %d %d %d) cost(%.4lf)\n",
         BATCH_SIZE, bn->inputs[0].dim[1], bn->inputs[0].dim[2],
         bn->inputs[0].dim[3], bn->runtime);
}

