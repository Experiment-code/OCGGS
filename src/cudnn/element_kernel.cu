#include "hip/hip_runtime.h"
/* Copyright 2019 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

__global__
void elementwise_kernel(int volume, OpType type,
                        const DATATYPE* x,
			const DATATYPE* y,
			DATATYPE* z)
{
  switch (type) {
    case OP_EW_SUB:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        z[i] = x[i] - y[i];
      }
      break;
    }
    case OP_EW_DIV:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        z[i] = x[i] / y[i];
      }
      break;
    }
    case OP_EW_EQUAL:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        z[i] = (x[i] == y[i]);
      }
      break;
    }
    case OP_EW_GREATER:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        z[i] = (x[i] > y[i]);
      }
      break;
    }
    case OP_EW_LESS:
    {
      CUDA_KERNEL_LOOP(i, volume)
      {
        z[i] = (x[i] < y[i]);
      }
      break;
    }
    default:
      assert(false);
  }
}

bool Element::has_cudnn_kernel(void) const
{
  switch (type) {
    case OP_EW_ADD:
    case OP_EW_MUL:
    case OP_EW_MAX:
    case OP_EW_MIN:
      return true;
    default:
      return false;
  }
}

void Element::map(void)
{
  if (has_cudnn_kernel()) {
    // create descriptors
    checkCUDNN(hipdnnCreateTensorDescriptor(&in1Tensor));
    checkCUDNN(hipdnnCreateTensorDescriptor(&in2Tensor));
    checkCUDNN(hipdnnCreateTensorDescriptor(&outTensor));
    checkCUDNN(hipdnnCreateOpTensorDescriptor(&opDesc));
    // set descriptors
    helperSetBroadcastableTensorDescriptor(inputs[0], outputs[0], in1Tensor);
    helperSetBroadcastableTensorDescriptor(inputs[1], outputs[0], in2Tensor);
    helperSetTensorDescriptor(outputs[0], outTensor);

    hipdnnOpTensorOp_t opType;
    switch (type) {
      case OP_EW_ADD:
        opType = HIPDNN_OP_TENSOR_ADD;
        break;
      case OP_EW_MUL:
        opType = HIPDNN_OP_TENSOR_MUL;
        break;
      case OP_EW_MAX:
        opType = HIPDNN_OP_TENSOR_MAX;
        break;
      case OP_EW_MIN:
        opType = HIPDNN_OP_TENSOR_MIN;
        break;
      default:
        fprintf(stderr, "Unsupported Elementwise Operator by cuDNN: %d\n", type);
        assert(false);
    }
    checkCUDNN(hipdnnSetOpTensorDescriptor(opDesc, opType, HIPDNN_DATA_FLOAT,
        HIPDNN_NOT_PROPAGATE_NAN));
  } else {
    // No preprocessing for our customized kernel
  }
  // allocate tensors
  size_t outputSize = sizeof(DATATYPE);
  for (int i = 0; i < outputs[0].numDim; i++)
    outputSize *= outputs[0].dim[i];
  checkCUDA(hipMalloc(&outputs[0].data_ptr, outputSize));
}

void Element::unmap(void)
{
  if (has_cudnn_kernel()) {
    checkCUDNN(hipdnnDestroyTensorDescriptor(in1Tensor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(in2Tensor));
    checkCUDNN(hipdnnDestroyTensorDescriptor(outTensor));
    checkCUDNN(hipdnnDestroyOpTensorDescriptor(opDesc));
  }
  checkCUDA(hipFree(outputs[0].data_ptr));
}

void Element::forward(bool block)
{
  if (has_cudnn_kernel()) {
    const float alpha = 1.0f;
    const float beta = 0.0f;
    checkCUDNN(hipdnnOpTensor(model->dnn, opDesc, &alpha, in1Tensor, inputs[0].data_ptr,
        &alpha, in2Tensor, inputs[1].data_ptr, &beta, outTensor, outputs[0].data_ptr));
  } else {
    elementwise_kernel<<<GET_BLOCKS(inputs[0].volume()), CUDA_NUM_THREADS>>>(
        inputs[0].volume(), type, (DATATYPE*)inputs[0].data_ptr, (DATATYPE*)inputs[1].data_ptr,
	(DATATYPE*)outputs[0].data_ptr);
  }
  if (block)
    checkCUDA(hipDeviceSynchronize());
}

void Model::measure_element_cost(Element* ele)
{
  // hipdnnOpTensor only supports OP_EW_ADD, OP_EW_MUL, OP_EW_MAX, OP_EW_MIN
  if (ele->has_cudnn_kernel()) {
    const float alpha = 1.0f;
    const float beta = 0.0f;
    helperSetBroadcastableTensorDescriptor(ele->inputs[0],
        ele->outputs[0], inputTensor);
    helperSetBroadcastableTensorDescriptor(ele->inputs[1],
        ele->outputs[0], biasTensor);
    helperSetTensorDescriptor(ele->outputs[0], outputTensor);
    hipdnnOpTensorOp_t opType;
    switch (ele->type) {
      case OP_EW_ADD:
        opType = HIPDNN_OP_TENSOR_ADD;
        break;
      case OP_EW_MUL:
        opType = HIPDNN_OP_TENSOR_MUL;
        break;
      case OP_EW_MAX:
        opType = HIPDNN_OP_TENSOR_MAX;
        break;
      case OP_EW_MIN:
        opType = HIPDNN_OP_TENSOR_MIN;
        break;
      default:
      {
        fprintf(stderr, "Unsupported Elementwise Operator by cuDNN: %d\n", ele->type);
        assert(false);
      }
    }
    checkCUDNN(hipdnnSetOpTensorDescriptor(opDesc, opType, HIPDNN_DATA_FLOAT,
        HIPDNN_NOT_PROPAGATE_NAN));
  
    checkCUDA(hipDeviceSynchronize());
    checkCUDA(hipEventRecord(startEvent));
    for (int i = 0; i < REPEAT_TIMES; i++) {
      checkCUDNN(hipdnnOpTensor(dnn, opDesc, &alpha, inputTensor, inputPtr,
          &alpha, biasTensor, filterPtr, &beta, outputTensor, outputPtr));
    }
    checkCUDA(hipEventRecord(endEvent));
    checkCUDA(hipEventSynchronize(endEvent));
    float milliseconds;
    hipEventElapsedTime(&milliseconds, startEvent, endEvent);
    ele->runtime = milliseconds / REPEAT_TIMES;
    if (print_cost)
      printf("  measure[Element]: i(%d %d %d %d) type(%d) cost(%.4lf)\n",
             ele->inputs[0].dim[0], ele->inputs[0].dim[1], ele->inputs[0].dim[2],
             ele->inputs[0].dim[3], ele->type, ele->runtime);
  } else {
    // Use our implementation to measure other elementwise operators
    checkCUDA(hipDeviceSynchronize());
    checkCUDA(hipEventRecord(startEvent));
    for (int i = 0; i < REPEAT_TIMES; i++) {
      elementwise_kernel<<<GET_BLOCKS(ele->inputs[0].volume()), CUDA_NUM_THREADS>>>(
          ele->inputs[0].volume(), ele->type, inputPtr, filterPtr, outputPtr);
    }
    checkCUDA(hipEventRecord(endEvent));
    checkCUDA(hipEventSynchronize(endEvent));
    float milliseconds;
    hipEventElapsedTime(&milliseconds, startEvent, endEvent);
    ele->runtime = milliseconds / REPEAT_TIMES;
    if (print_cost)
      printf("  measure[Element]: i(%d %d %d %d) type(%d) cost(%.4lf)\n",
             ele->inputs[0].dim[0], ele->inputs[0].dim[1], ele->inputs[0].dim[2],
             ele->inputs[0].dim[3], ele->type, ele->runtime);
  }
}

